#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

#include "gpumat.h"
#include "cuda_common.h"
#include "common_types.h"

#include "common_devices.h"

using namespace gpumat;

///////// begin internal namespace ///////////////

namespace gpumat{

namespace internal{

template< typename T >
__global__ void im2cols(Mtx X, ct::Size szA0, int channels, ct::Size szW, int stride, Mtx Res, ct::Size szOut)
{
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	if(col < szOut.width && row < szOut.height){
		int x0 = col * stride;
		int y0 = row * stride;
		int row2 = row * szOut.width + col;

		int szA0area = szA0.width * szA0.height;
		int szWarea = szW.width * szW.height;

		T *dX = (T*)X.data;
		T *dR = (T*)Res.data;
		for(int c = 0; c < channels; ++c){
			T *dXi = &dX[c * szA0area];

			for(int a = 0; a < szW.height; ++a){
				for(int b = 0; b < szW.width; ++b){
					int col2 = c * szWarea + (a * szW.width + b);
					dR[row2 * Res.cols + col2] = dXi[(y0 + a) * szA0.width + (x0 + b)];
				}
			}
		}
	}
}

}

}

extern "C"
void cuda_im2cols(const gpumat::GpuMat &X,
				  const ct::Size &szA0,
				  int channels,
				  const ct::Size &szW,
				  int stride,
				  gpumat::GpuMat &Res,
				  ct::Size &szOut)
{
	int x1 = szOut.width / BLOCKSIZE + 1;
	int x2 = szOut.height / BLOCKSIZE + 1;

	dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE, BLOCKSIZE);

	switch (X.type) {
		case GPU_DOUBLE:
			internal::im2cols<double> <<<dimGrid, dimBlock>>>(X, szA0, channels, szW, stride, Res, szOut);
			break;
		case GPU_FLOAT:
			internal::im2cols<float> <<<dimGrid, dimBlock>>>(X, szA0, channels, szW, stride, Res, szOut);
			break;
	}

}
